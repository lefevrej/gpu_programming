#include <stdio.h>

int main() {
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount( &count );
    for(int i=0; i< count; i++) {
        hipGetDeviceProperties( &prop, i );
        printf( "   ---General Information for device%d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Computecapability:  %d.%d\n", prop.major, prop.minor);
        printf( "Clockrate:  %d\n", prop.clockRate);
        printf( "   ---Memory Information for device%d ---\n", i );
        printf( "Total global mem:  %lu\n", prop.totalGlobalMem);
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem);
        printf( "Max mem pitch:  %ld\n", prop.memPitch);
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment);printf( "   ---MP Information for device%d ---\n", i );
        printf( "Multiprocessorcount:  %d\n",  prop.multiProcessorCount);printf( "Sharedmem per mp:  %ld\n", prop.sharedMemPerBlock);
        printf( "Registersper mp:  %d\n", prop.regsPerBlock);
        printf( "Threads in warp:  %d\n", prop.warpSize);
        printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock);
        printf( "Max thread dimensions:  (%d, %d, %d)\n",prop.maxThreadsDim[0], prop.maxThreadsDim[1],prop.maxThreadsDim[2] );
        printf( "Max griddimensions:  (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1],prop.maxGridSize[2] );
        printf( "\n" );
    }
    return 0;
}